
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 610)
#include <sm_61_intrinsics.h>
#endif
extern "C" __global__ void fused_divide_add_round_cast_clip_cast_23_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 3.052289e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_22_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 4.320321e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__23_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[4];
  __shared__ int pad_data_shared[324];
  __shared__ int placeholder_shared[576];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
        ((int*)((signed char*)pad_data_shared + ((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + (((((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) - 32))))[0] : (int)(int)0);
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 6) {
            ((int*)((signed char*)placeholder_shared + (((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 63; ++ic_chunk_outer_outer) {
    __syncthreads();
    if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
      if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
          ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 648) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 392) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) + 360))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 6) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (ic_chunk_outer_outer * 288)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    #pragma unroll
    for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
      #pragma unroll
      for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 648) + (ic_chunk_inner * 324)) + (((int)threadIdx.y) * 36)) + (kh_inner * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
    #pragma unroll
    for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_inner1 * 324) + (((int)threadIdx.y) * 36)) + (kh_inner1 * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 648))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(oc_block1)]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((signed char*)T_cast)[((((((((int)blockIdx.y) * 784) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(ax4)]) * 1.454542e-02f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 9.007953e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
  }
}

extern "C" __global__ void fused_cast_clip_cast_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))])), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_nn_global_avg_pool2d_2_kernel0(void* __restrict__ placeholder, void* __restrict__ tensor) {
  float tensor1[4];
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    tensor1[(ax4)] = 0.000000e+00f;
    for (int rv0 = 0; rv0 < 7; ++rv0) {
      for (int rv1 = 0; rv1 < 7; ++rv1) {
        if (((int)threadIdx.y) < 1) {
          tensor1[(ax4)] = (tensor1[(ax4)] + ((float*)placeholder)[(((((((((int)threadIdx.y) * 25088) + (((int)blockIdx.x) * 1568)) + (((int)threadIdx.x) * 196)) + (rv0 * 28)) + (rv1 * 4)) + ax4))]);
        }
      }
    }
  }
  for (int ax41 = 0; ax41 < 4; ++ax41) {
    if (((int)threadIdx.y) < 1) {
      ((float*)tensor)[(((((((int)threadIdx.y) * 512) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 4)) + ax41))] = (tensor1[(ax41)] * 2.040816e-02f);
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__16_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int placeholder_shared[576];
  __shared__ int pad_data_shared[360];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 11) {
            ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 7; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 13) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + ((((((((ic_chunk_outer_outer * 25088) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) - 228))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 11) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (ic_chunk_outer_outer * 288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
          #pragma unroll
          for (int oh = 0; oh < 4; ++oh) {
            #pragma unroll
            for (int oc_block = 0; oc_block < 4; ++oc_block) {
              compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner * 720) + (oh * 120)) + (kh_inner * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
            }
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 13) {
          ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + (((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) + 175388))))[0] : (int)(int)0);
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
        #pragma unroll
        for (int oh1 = 0; oh1 < 4; ++oh1) {
          #pragma unroll
          for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
            compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 720) + (oh1 * 120)) + (kh_inner1 * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(((oh1 * 4) + oc_block1))]);
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((signed char*)T_cast)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 9.864906e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 6.070983e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_nn_pad_layout_transform_kernel0(void* __restrict__ T_layout_trans, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_layout_trans)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = (((((int)threadIdx.x) & 3) < 3) ? (signed char)((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((((int)threadIdx.x) & 3) * 50176) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)))] * 5.070899e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128))) : (signed char)(signed char)0);
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_21_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int pad_data_shared[1320];
  __shared__ int placeholder_shared[256];
  for (int oh_init = 0; oh_init < 2; ++oh_init) {
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
      compute[((((oh_init * 4) + oc_block_init) + 8))] = 0;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 660) {
        ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 165) * 12544) + (((int)blockIdx.x) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 165) / 55) * 224)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 55) * 4)))))[0];
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 32) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 128) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 5) {
          if ((((((int)blockIdx.y) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 7)) + (((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) >> 2)) < 32) {
              ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 1792)) + ((((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) >> 2) * 256)) + ((((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
          }
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 3; ++ic_chunk_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 660) {
          ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 2640) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 50176) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 165) * 12544)) + (((int)blockIdx.x) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 165) / 55) * 224)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 55) * 4)) + 50176))))[0];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 32) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 128) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 5) {
            if ((((((int)blockIdx.y) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 7)) + (((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) >> 2)) < 32) {
                ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 512) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 1792)) + ((((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) >> 2) * 256)) + (ic_chunk_outer_outer * 64)) + ((((((int)threadIdx.z) * 7) + (((int)threadIdx.x) >> 2)) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64))))[0];
            }
          }
        }
      }
    }
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      for (int oh = 0; oh < 2; ++oh) {
        for (int oc_block = 0; oc_block < 4; ++oc_block) {
          compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer & 1) * 2640) + (ic_chunk_inner * 660)) + (oh * 440)) + (((int)threadIdx.x) * 8)))))[0], ((int*)((signed char*)placeholder_shared + ((((((ic_chunk_outer_outer & 1) * 512) + (((int)threadIdx.z) * 64)) + (ic_chunk_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
          compute[((((oh * 4) + oc_block) + 8))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer & 1) * 2640) + (ic_chunk_inner * 660)) + (oh * 440)) + (((int)threadIdx.x) * 8)))))[0], ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer & 1) * 512) + (((int)threadIdx.z) * 64)) + (ic_chunk_inner * 16)) + (oc_block * 4)) + 256))))[0], compute[((((oh * 4) + oc_block) + 8))]);
        }
      }
    }
  }
  __syncthreads();
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 4; ++ic_chunk_inner1) {
    for (int oh1 = 0; oh1 < 2; ++oh1) {
      for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
        compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((ic_chunk_inner1 * 660) + (oh1 * 440)) + (((int)threadIdx.x) * 8)) + 2640))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 64) + (ic_chunk_inner1 * 16)) + (oc_block1 * 4)) + 512))))[0], compute[(((oh1 * 4) + oc_block1))]);
        compute[((((oh1 * 4) + oc_block1) + 8))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((ic_chunk_inner1 * 660) + (oh1 * 440)) + (((int)threadIdx.x) * 8)) + 2640))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 64) + (ic_chunk_inner1 * 16)) + (oc_block1 * 4)) + 768))))[0], compute[((((oh1 * 4) + oc_block1) + 8))]);
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_relu)[(((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.x) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 2.086217e-04f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))]) + ((float*)placeholder3)[(((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.x) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)threadIdx.x) * 4)) + ax4))])), (float)(0.000000e+00f));
      ((float*)T_relu)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.x) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 12544))] = max((float)((((((float)compute[((((ax2_inner_inner_inner * 4) + ax4) + 8))]) * 2.086217e-04f) + ((float*)placeholder2)[(((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4) + 16))]) + ((float*)placeholder3)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.x) * 224)) + (ax2_inner_inner_inner * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 12544))])), (float)(0.000000e+00f));
    }
  }
}

extern "C" __global__ void fused_nn_dense_add_2_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_add, void* __restrict__ placeholder2) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (((float*)placeholder)[(((k_outer * 64) + ((int)threadIdx.x)))] * ((float*)placeholder1)[((((((int)blockIdx.x) * 512) + (k_outer * 64)) + ((int)threadIdx.x)))]));
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = ((volatile float*)red_buf0)[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    ((float*)T_add)[(((int)blockIdx.x))] = (T_dense[(0)] + ((float*)placeholder2)[(((int)blockIdx.x))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__21_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[8];
  __shared__ int pad_data_shared[512];
  __shared__ int placeholder_shared[2304];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
    compute[((oc_block_init + 4))] = 0;
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 19) {
          ((int*)((signed char*)pad_data_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + ((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 60))))[0] : (int)(int)0);
      }
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 83) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 15; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 19) {
            ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1024) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784)) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3076))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 83) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 4608) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + (ic_chunk_outer_outer * 576)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 576))))[0];
        }
      }
    }
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
            compute[((oc_block + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)) + 64))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[((oc_block + 4))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 4; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1024))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[(oc_block1)]);
          compute[((oc_block1 + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1088))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[((oc_block1 + 4))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((signed char*)T_cast)[((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(ax4)]) * 6.764178e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))])), (float)(0.000000e+00f)) * 4.371008e+00f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    ((signed char*)T_cast)[(((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 56))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[((ax4 + 4))]) * 6.764178e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))])), (float)(0.000000e+00f)) * 4.371008e+00f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_2_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2) {
  int compute[32];
  __shared__ int placeholder_shared[896];
  __shared__ int pad_data_shared[259];
  for (int oc_chunk_init = 0; oc_chunk_init < 2; ++oc_chunk_init) {
    for (int oh_init = 0; oh_init < 4; ++oh_init) {
      for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
        compute[((((oc_chunk_init * 16) + (oh_init * 4)) + oc_block_init))] = 0;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) < 112) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) < 448) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 28) {
            ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) / 7) * 784) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) % 7) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int kh_outer_outer = 0; kh_outer_outer < 6; ++kh_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) < 259) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 17) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.x) * 4)))))[0] = (((((3 <= ((((((int)blockIdx.x) / 7) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) / 37)) + kh_outer_outer)) && (((((((int)blockIdx.x) / 7) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) / 37)) + kh_outer_outer) < 227)) && (3 <= (((((int)blockIdx.x) % 7) * 32) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37)) < 227)) ? (int)((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.x) / 7) * 7168) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) / 37) * 896)) + (kh_outer_outer * 896)) + ((((int)blockIdx.x) % 7) * 128)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37) * 4)) - 2700))))[0] : (int)(int)0);
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) < 112) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) < 448) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 28) {
              ((int*)((signed char*)placeholder_shared + (((((((kh_outer_outer + 1) & 1) * 1792) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 512)) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + ((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) / 7) * 784) + (kh_outer_outer * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 32) + (((int)threadIdx.z) * 4)) + (((int)threadIdx.x) >> 2)) % 7) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 112))))[0];
          }
        }
      }
    }
    __syncthreads();
    for (int kw_inner = 0; kw_inner < 7; ++kw_inner) {
      for (int oc_chunk = 0; oc_chunk < 2; ++oc_chunk) {
        for (int oh = 0; oh < 4; ++oh) {
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[((((oc_chunk * 16) + (oh * 4)) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((oh * 296) + (((int)threadIdx.x) * 8)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((kh_outer_outer & 1) * 1792) + (((int)threadIdx.z) * 224)) + (oc_chunk * 112)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[((((oc_chunk * 16) + (oh * 4)) + oc_block))]);
          }
        }
      }
    }
  }
  __syncthreads();
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) < 259) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 17) {
          ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 512) + (((int)threadIdx.z) * 64)) + (((int)threadIdx.x) * 4)))))[0] = (((((((((int)blockIdx.x) / 7) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) / 37)) < 221) && (3 <= (((((int)blockIdx.x) % 7) * 32) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37)))) && ((((((int)blockIdx.x) % 7) * 32) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37)) < 227)) ? (int)((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.x) / 7) * 7168) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) / 37) * 896)) + ((((int)blockIdx.x) % 7) * 128)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 128) + (((int)threadIdx.z) * 16)) + ((int)threadIdx.x)) % 37) * 4)) + 2676))))[0] : (int)(int)0);
      }
    }
  }
  __syncthreads();
  for (int kw_inner1 = 0; kw_inner1 < 7; ++kw_inner1) {
    for (int oc_chunk1 = 0; oc_chunk1 < 2; ++oc_chunk1) {
      for (int oh1 = 0; oh1 < 4; ++oh1) {
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[((((oc_chunk1 * 16) + (oh1 * 4)) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((oh1 * 296) + (((int)threadIdx.x) * 8)) + (kw_inner1 * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 224) + (oc_chunk1 * 112)) + (kw_inner1 * 16)) + (oc_block1 * 4)))))[0], compute[((((oc_chunk1 * 16) + (oh1 * 4)) + oc_block1))]);
        }
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
      for (int ax4 = 0; ax4 < 4; ++ax4) {
        ((float*)T_relu)[((((((((((int)threadIdx.z) * 100352) + (ax1_inner_inner_inner * 50176)) + ((((int)blockIdx.x) / 7) * 1792)) + (ax2_inner_inner_inner * 448)) + ((((int)blockIdx.x) % 7) * 64)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)(((((float)compute[((((ax1_inner_inner_inner * 16) + (ax2_inner_inner_inner * 4)) + ax4))]) * 3.420994e-05f) + ((float*)placeholder2)[((((((int)threadIdx.z) * 8) + (ax1_inner_inner_inner * 4)) + ax4))])), (float)(0.000000e+00f));
      }
    }
  }
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_20_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 2.461749e+00f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_27_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 3.938798e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_22_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int placeholder_shared[576];
  __shared__ int pad_data_shared[360];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 11) {
            ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 7; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 13) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + ((((((((ic_chunk_outer_outer * 25088) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) - 228))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 11) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (ic_chunk_outer_outer * 288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
          #pragma unroll
          for (int oh = 0; oh < 4; ++oh) {
            #pragma unroll
            for (int oc_block = 0; oc_block < 4; ++oc_block) {
              compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner * 720) + (oh * 120)) + (kh_inner * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
            }
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 13) {
          ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + (((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) + 175388))))[0] : (int)(int)0);
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
        #pragma unroll
        for (int oh1 = 0; oh1 < 4; ++oh1) {
          #pragma unroll
          for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
            compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 720) + (oh1 * 120)) + (kh_inner1 * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(((oh1 * 4) + oc_block1))]);
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_relu)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((float*)placeholder2)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] + ((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 1.386576e-04f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))]))), (float)(0.000000e+00f));
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__17_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int placeholder_shared[576];
  __shared__ int pad_data_shared[360];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 11) {
            ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 7; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 13) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + ((((((((ic_chunk_outer_outer * 25088) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) - 228))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 11) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (ic_chunk_outer_outer * 288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
          #pragma unroll
          for (int oh = 0; oh < 4; ++oh) {
            #pragma unroll
            for (int oc_block = 0; oc_block < 4; ++oc_block) {
              compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner * 720) + (oh * 120)) + (kh_inner * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
            }
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 13) {
          ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + (((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) + 175388))))[0] : (int)(int)0);
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
        #pragma unroll
        for (int oh1 = 0; oh1 < 4; ++oh1) {
          #pragma unroll
          for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
            compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 720) + (oh1 * 120)) + (kh_inner1 * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(((oh1 * 4) + oc_block1))]);
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((signed char*)T_cast)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 6.221216e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 4.040347e+00f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_24_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 3.578871e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_25_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 2.879392e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_28_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 6272) {
    if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 25088) {
      ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 2.156147e+00f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_17_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[4];
  __shared__ int pad_data_shared[104];
  __shared__ int placeholder_shared[256];
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
  }
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 52) {
      ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) / 13) * 784) + (((int)blockIdx.x) * 112)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) % 13) * 4)))))[0];
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) < 128) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 19) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) >> 4) * 64) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) >> 2)) & 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((int)blockIdx.y) * 8192) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) >> 4) * 1024)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) >> 2)) & 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 15; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) < 52) {
        ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 208) + (((int)threadIdx.z) * 28)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 3136) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) / 13) * 784)) + (((int)blockIdx.x) * 112)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) % 13) * 4)) + 3136))))[0];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) < 128) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 19) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 512) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) >> 4) * 64)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) >> 2)) & 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 8192) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.x)) >> 4) * 1024)) + (ic_chunk_outer_outer * 64)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) >> 2)) & 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) & 3) * 4)) + 64))))[0];
        }
      }
    }
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      for (int oc_block = 0; oc_block < 4; ++oc_block) {
        compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((ic_chunk_outer_outer & 1) * 208) + (ic_chunk_inner * 52)) + (((int)threadIdx.x) * 8)))))[0], ((int*)((signed char*)placeholder_shared + ((((((ic_chunk_outer_outer & 1) * 512) + (((int)threadIdx.z) * 64)) + (ic_chunk_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
      }
    }
  }
  __syncthreads();
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 4; ++ic_chunk_inner1) {
    for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
      compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((ic_chunk_inner1 * 52) + (((int)threadIdx.x) * 8)) + 208))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 64) + (ic_chunk_inner1 * 16)) + (oc_block1 * 4)) + 512))))[0], compute[(oc_block1)]);
    }
  }
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((float*)T_relu)[((((((((int)blockIdx.y) * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.x) * 28)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((((float)compute[(ax4)]) * 5.475191e-02f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))]) + ((float*)placeholder3)[((((((((int)blockIdx.y) * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.x) * 28)) + (((int)threadIdx.x) * 4)) + ax4))])), (float)(0.000000e+00f));
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_8_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_add, void* __restrict__ placeholder2) {
  int compute[4];
  __shared__ int pad_data_shared[324];
  __shared__ int placeholder_shared[576];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
        ((int*)((signed char*)pad_data_shared + ((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + (((((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) - 32))))[0] : (int)(int)0);
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 6) {
            ((int*)((signed char*)placeholder_shared + (((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 63; ++ic_chunk_outer_outer) {
    __syncthreads();
    if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
      if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
          ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 648) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 392) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) + 360))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 6) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (ic_chunk_outer_outer * 288)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    #pragma unroll
    for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
      #pragma unroll
      for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 648) + (ic_chunk_inner * 324)) + (((int)threadIdx.y) * 36)) + (kh_inner * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
    #pragma unroll
    for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_inner1 * 324) + (((int)threadIdx.y) * 36)) + (kh_inner1 * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 648))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(oc_block1)]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((float*)T_add)[((((((((int)blockIdx.y) * 784) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)) + ax4))] = ((((float)compute[(ax4)]) * 1.372629e-04f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__22_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[4];
  __shared__ int pad_data_shared[90];
  __shared__ int placeholder_shared[576];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
  }
  if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 45) {
    if (((int)threadIdx.z) < 7) {
        ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.z) * 28) + (((int)threadIdx.y) * 4)))))[0] = (((3 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.y))) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) % 3)))) ? (int)((int*)((signed char*)placeholder + ((((((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) / 3) * 56) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) % 3) * 4)) - 60))))[0] : (int)(int)0);
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 288) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 42) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) / 36) * 9216)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 63; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 45) {
      if (((int)threadIdx.z) < 7) {
          ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 180) + (((int)threadIdx.z) * 28)) + (((int)threadIdx.y) * 4)))))[0] = (((3 <= ((((int)threadIdx.z) * 7) + ((int)threadIdx.y))) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) % 3)))) ? (int)((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 784) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) / 3) * 56)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) % 3) * 4)) + 724))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 42) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) / 36) * 9216)) + (ic_chunk_outer_outer * 144)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 14) + (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) & 3) * 4)) + 144))))[0];
        }
      }
    }
    #pragma unroll
    for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int oc_block = 0; oc_block < 4; ++oc_block) {
          compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer & 1) * 180) + (((int)threadIdx.y) * 24)) + (kh_inner * 12)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
        compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((int)threadIdx.y) * 24) + (kh_inner1 * 12)) + (kw_inner1 * 4)) + 180))))[0], ((int*)((signed char*)placeholder_shared + ((((((((int)threadIdx.z) * 144) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(oc_block1)]);
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((signed char*)T_cast)[((((((((int)blockIdx.y) * 1568) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(ax4)]) * 6.869003e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 7.446812e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__18_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[8];
  __shared__ int pad_data_shared[580];
  __shared__ int placeholder_shared[2304];
  for (int oh_init = 0; oh_init < 2; ++oh_init) {
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  for (int ic_chunk_outer = 0; ic_chunk_outer < 4; ++ic_chunk_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 580) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 16) + ((int)threadIdx.z)) < 42) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 896) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = (((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 145) / 29))) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 29)))) ? (int)((int*)((signed char*)placeholder + ((((((((ic_chunk_outer * 50176) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 145) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 145) / 29) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 29) * 4)) - 228))))[0] : (int)(int)0);
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 2304) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 16) + ((int)threadIdx.z)) < 165) {
            ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 144) * 2304)) + (ic_chunk_outer * 576)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
    __syncthreads();
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
          for (int oh = 0; oh < 2; ++oh) {
            for (int oc_block = 0; oc_block < 4; ++oc_block) {
              compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner * 580) + (oh * 232)) + (kh_inner * 116)) + (((int)threadIdx.x) * 8)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((int)threadIdx.z) * 576) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
            }
          }
        }
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((signed char*)T_cast)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 3136)) + ((((int)blockIdx.x) >> 1) * 224)) + (ax2_inner_inner_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 6.121386e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 64) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 5.916168e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_7_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_add, void* __restrict__ placeholder2) {
  int compute[8];
  __shared__ int pad_data_shared[512];
  __shared__ int placeholder_shared[2304];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
    compute[((oc_block_init + 4))] = 0;
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 19) {
          ((int*)((signed char*)pad_data_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + ((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 60))))[0] : (int)(int)0);
      }
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 83) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 15; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 19) {
            ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1024) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784)) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3076))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 83) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 4608) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + (ic_chunk_outer_outer * 576)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 576))))[0];
        }
      }
    }
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
            compute[((oc_block + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)) + 64))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[((oc_block + 4))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 4; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1024))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[(oc_block1)]);
          compute[((oc_block1 + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1088))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[((oc_block1 + 4))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((float*)T_add)[((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = ((((float)compute[(ax4)]) * 9.887036e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))]);
    ((float*)T_add)[(((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 56))] = ((((float)compute[((ax4 + 4))]) * 9.887036e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_19_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[8];
  __shared__ int pad_data_shared[432];
  __shared__ int placeholder_shared[1024];
  for (int oc_chunk_init = 0; oc_chunk_init < 2; ++oc_chunk_init) {
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oc_chunk_init * 4) + oc_block_init))] = 0;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 216) {
        ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 27) * 3136) + (((int)blockIdx.x) * 224)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 27) * 4)))))[0];
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 512) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 37) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) >> 5) * 128) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) & 7) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((int)blockIdx.y) * 8192) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) >> 5) * 512)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) & 7) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 3; ++ic_chunk_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 216) {
          ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 864) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 25088) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 27) * 3136)) + (((int)blockIdx.x) * 224)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 27) * 4)) + 25088))))[0];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 512) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 37) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 2048) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) >> 5) * 128)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) & 7) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 8192) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) >> 5) * 512)) + (ic_chunk_outer_outer * 128)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) & 7) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 128))))[0];
        }
      }
    }
    for (int ic_chunk_inner = 0; ic_chunk_inner < 8; ++ic_chunk_inner) {
      for (int oc_chunk = 0; oc_chunk < 2; ++oc_chunk) {
        for (int oc_block = 0; oc_block < 4; ++oc_block) {
          compute[(((oc_chunk * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((ic_chunk_outer_outer & 1) * 864) + (ic_chunk_inner * 108)) + (((int)threadIdx.x) * 8)))))[0], ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer & 1) * 2048) + (((int)threadIdx.z) * 256)) + (oc_chunk * 128)) + (ic_chunk_inner * 16)) + (oc_block * 4)))))[0], compute[(((oc_chunk * 4) + oc_block))]);
        }
      }
    }
  }
  __syncthreads();
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 8; ++ic_chunk_inner1) {
    for (int oc_chunk1 = 0; oc_chunk1 < 2; ++oc_chunk1) {
      for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
        compute[(((oc_chunk1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((ic_chunk_inner1 * 108) + (((int)threadIdx.x) * 8)) + 864))))[0], ((int*)((signed char*)placeholder_shared + ((((((((int)threadIdx.z) * 256) + (oc_chunk1 * 128)) + (ic_chunk_inner1 * 16)) + (oc_block1 * 4)) + 2048))))[0], compute[(((oc_chunk1 * 4) + oc_block1))]);
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_relu)[(((((((((int)blockIdx.y) * 12544) + (((int)threadIdx.z) * 1568)) + (ax1_inner_inner_inner * 784)) + (((int)blockIdx.x) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((((float)compute[(((ax1_inner_inner_inner * 4) + ax4))]) * 2.299786e-02f) + ((float*)placeholder2)[(((((((int)blockIdx.y) * 64) + (((int)threadIdx.z) * 8)) + (ax1_inner_inner_inner * 4)) + ax4))]) + ((float*)placeholder3)[(((((((((int)blockIdx.y) * 12544) + (((int)threadIdx.z) * 1568)) + (ax1_inner_inner_inner * 784)) + (((int)blockIdx.x) * 56)) + (((int)threadIdx.x) * 4)) + ax4))])), (float)(0.000000e+00f));
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_divide_add_round_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_round, void* __restrict__ placeholder2, void* __restrict__ placeholder3, void* __restrict__ placeholder4) {
  int compute[16];
  __shared__ int placeholder_shared[576];
  __shared__ int pad_data_shared[360];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 11) {
            ((int*)((signed char*)placeholder_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 7; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 13) {
            ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + ((((((((ic_chunk_outer_outer * 25088) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) - 228))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) < 72) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 288) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 11) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 448)) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = ((int*)((signed char*)placeholder + (((((((((int)blockIdx.y) * 9216) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) / 18) * 2304)) + (ic_chunk_outer_outer * 288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.x) >> 2)) % 18) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
          #pragma unroll
          for (int oh = 0; oh < 4; ++oh) {
            #pragma unroll
            for (int oc_block = 0; oc_block < 4; ++oc_block) {
              compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner * 720) + (oh * 120)) + (kh_inner * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
            }
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) < 360) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 13) {
          ((int*)((signed char*)pad_data_shared + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 448) + (((int)threadIdx.z) * 112)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30)) < 57)) && (1 <= (((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)))) && ((((((int)blockIdx.x) & 1) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30)) < 57)) ? (int)((int*)((signed char*)placeholder1 + (((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) / 180) * 12544) + ((((int)blockIdx.x) >> 1) * 896)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 180) / 30) * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 28)) + ((int)threadIdx.x)) % 30) * 4)) + 175388))))[0] : (int)(int)0);
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
        #pragma unroll
        for (int oh1 = 0; oh1 < 4; ++oh1) {
          #pragma unroll
          for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
            compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 720) + (oh1 * 120)) + (kh_inner1 * 120)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(((oh1 * 4) + oc_block1))]);
          }
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_round)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = roundf(((max((float)((((float*)placeholder2)[((((((((((int)blockIdx.y) * 50176) + (((int)threadIdx.z) * 12544)) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner_inner_inner * 224)) + ((((int)blockIdx.x) & 1) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] + ((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 3.298299e-02f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))]))), (float)(0.000000e+00f)) * 2.426362e+01f) + ((float*)placeholder4)[(0)]));
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_20_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int pad_data_shared[192];
  __shared__ int placeholder_shared[288];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
    if (((int)threadIdx.z) < 7) {
        ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((((int)blockIdx.x) >> 1) * 448) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 116))))[0] : (int)(int)0);
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 31; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
      if (((int)threadIdx.z) < 7) {
          ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 384) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + ((((int)blockIdx.x) >> 1) * 448)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3020))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 21) {
            ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + (ic_chunk_outer_outer * 144)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int oh = 0; oh < 4; ++oh) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 384) + (oh * 64)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 21) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 4464))))[0];
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int oh1 = 0; oh1 < 4; ++oh1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((oh1 * 64) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 384))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)))))[0], compute[(((oh1 * 4) + oc_block1))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_relu)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + ((((int)blockIdx.x) >> 1) * 448)) + (ax2_inner_inner_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((float*)placeholder2)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + ((((int)blockIdx.x) >> 1) * 448)) + (ax2_inner_inner_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] + ((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 1.310110e-04f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))]))), (float)(0.000000e+00f));
    }
  }
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_26_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 3.980142e+01f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_18_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[8];
  __shared__ int pad_data_shared[512];
  __shared__ int placeholder_shared[2304];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
    compute[((oc_block_init + 4))] = 0;
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 19) {
          ((int*)((signed char*)pad_data_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + ((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 60))))[0] : (int)(int)0);
      }
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.y)) < 83) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 15; ++ic_chunk_outer_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 256) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 19) {
            ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1024) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 64)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)))))[0] = (((((1 <= ((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3))) && (((((int)blockIdx.x) * 2) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3)) < 15)) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15))) && ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) < 15)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) >> 6) * 784)) + (((int)blockIdx.x) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_outer_fused_outer_outer_outer1 * 7) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 4)) & 3) * 56)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3076))))[0] : (int)(int)0);
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 1152) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.y)) < 83) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 4608) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 144) * 9216)) + (ic_chunk_outer_outer * 576)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 144) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 576))))[0];
        }
      }
    }
    #pragma unroll
    for (int ic_chunk_inner = 0; ic_chunk_inner < 4; ++ic_chunk_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
            compute[((oc_block + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 1024) + (ic_chunk_inner * 256)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)) + 64))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 4608) + (((int)threadIdx.y) * 576)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[((oc_block + 4))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 4; ++ic_chunk_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1024))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[(oc_block1)]);
          compute[((oc_block1 + 4))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((ic_chunk_inner1 * 256) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 1088))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.y) * 576) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 4608))))[0], compute[((oc_block1 + 4))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((float*)T_relu)[((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((float*)placeholder2)[((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4))] + ((((float)compute[(ax4)]) * 2.007595e-02f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))]))), (float)(0.000000e+00f));
    ((float*)T_relu)[(((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 56))] = max((float)((((float*)placeholder2)[(((((((((int)blockIdx.y) * 6272) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (((int)threadIdx.x) * 4)) + ax4) + 56))] + ((((float)compute[((ax4 + 4))]) * 2.007595e-02f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.y) * 4)) + ax4))]))), (float)(0.000000e+00f));
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__19_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[16];
  __shared__ int pad_data_shared[192];
  __shared__ int placeholder_shared[288];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
    if (((int)threadIdx.z) < 7) {
        ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((((int)blockIdx.x) >> 1) * 448) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 116))))[0] : (int)(int)0);
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 31; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
      if (((int)threadIdx.z) < 7) {
          ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 384) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + ((((int)blockIdx.x) >> 1) * 448)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3020))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 21) {
            ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + (ic_chunk_outer_outer * 144)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int oh = 0; oh < 4; ++oh) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 384) + (oh * 64)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 21) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 4464))))[0];
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int oh1 = 0; oh1 < 4; ++oh1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((oh1 * 64) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 384))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)))))[0], compute[(((oh1 * 4) + oc_block1))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((signed char*)T_cast)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + ((((int)blockIdx.x) >> 1) * 448)) + (ax2_inner_inner_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 1.082206e-04f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))])), (float)(0.000000e+00f)) * 5.238856e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_nn_max_pool2d_2_kernel0(void* __restrict__ placeholder, void* __restrict__ tensor) {
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int rv = 0; rv < 3; ++rv) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      tensor_local[(0)] = max((float)(tensor_local[(0)]), (float)((((1 <= ((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 3136) / 56) * 2) + rv)) && (1 <= (((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 56) * 2) + rv1))) ? (float)((float*)placeholder)[((((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 56) * 896) + (rv * 448)) + ((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 56) * 8)) + (rv1 * 4)) + (((int)threadIdx.x) & 3)) - 452))] : (float)-3.402823e+38f)));
    }
  }
  ((float*)tensor)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_nn_relu_divide_add_round_cast_cl_16286804049590463984__20_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_cast, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[8];
  __shared__ int pad_data_shared[290];
  __shared__ int placeholder_shared[1152];
  #pragma unroll
  for (int oh_init = 0; oh_init < 2; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 145) {
    if (((int)threadIdx.y) < 11) {
        ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.y) * 56) + (((int)threadIdx.x) * 4)))))[0] = (((1 <= ((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) / 29))) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29))) ? (int)((int*)((signed char*)placeholder + (((((((int)blockIdx.x) * 448) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) / 29) * 112)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29) * 4)) - 116))))[0] : (int)(int)0);
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 576) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 16) + ((int)threadIdx.y)) < 42) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 56) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 31; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 145) {
      if (((int)threadIdx.y) < 11) {
          ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 580) + (((int)threadIdx.y) * 56)) + (((int)threadIdx.x) * 4)))))[0] = (((1 <= ((((int)blockIdx.x) * 4) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) / 29))) && (1 <= (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29))) ? (int)((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 3136) + (((int)blockIdx.x) * 448)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) / 29) * 112)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) % 29) * 4)) + 3020))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 576) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 16) + ((int)threadIdx.y)) < 42) {
            ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 2304) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + (ic_chunk_outer_outer * 144)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 224) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 56) + (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 144))))[0];
        }
      }
    }
    #pragma unroll
    for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
      #pragma unroll
      for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
        #pragma unroll
        for (int oh = 0; oh < 2; ++oh) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 580) + (oh * 232)) + (kh_inner * 116)) + (((int)threadIdx.x) * 8)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer & 1) * 2304) + (((int)threadIdx.y) * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
    #pragma unroll
    for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
      #pragma unroll
      for (int oh1 = 0; oh1 < 2; ++oh1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((oh1 * 232) + (kh_inner1 * 116)) + (((int)threadIdx.x) * 8)) + (kw_inner1 * 4)) + 580))))[0], ((int*)((signed char*)placeholder_shared + ((((((((int)threadIdx.y) * 144) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 2304))))[0], compute[(((oh1 * 4) + oc_block1))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 2; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((signed char*)T_cast)[(((((((((int)blockIdx.y) * 12544) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.x) * 112)) + (ax2_inner_inner_inner * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = ((signed char)max((int)(min((int)(((int)roundf(((max((float)(((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 6.005610e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 64) + (((int)threadIdx.y) * 4)) + ax4))])), (float)(0.000000e+00f)) * 5.611094e+01f) + ((float*)placeholder3)[(0)])))), (int)(127))), (int)(-128)));
    }
  }
}

extern "C" __global__ void fused_divide_add_round_cast_clip_cast_21_kernel0(void* __restrict__ T_cast, void* __restrict__ placeholder, void* __restrict__ placeholder1) {
  ((signed char*)T_cast)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] = ((signed char)max((int)(min((int)(((int)roundf(((((float*)placeholder)[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)))] * 1.799620e+00f) + ((float*)placeholder1)[(0)])))), (int)(127))), (int)(-128)));
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_add_nn_relu_16_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_relu, void* __restrict__ placeholder2, void* __restrict__ placeholder3) {
  int compute[4];
  __shared__ int pad_data_shared[324];
  __shared__ int placeholder_shared[576];
  #pragma unroll
  for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
    compute[(oc_block_init)] = 0;
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
        ((int*)((signed char*)pad_data_shared + ((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + (((((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) - 32))))[0] : (int)(int)0);
    }
  }
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
    if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 4) + ((int)threadIdx.z)) < 6) {
            ((int*)((signed char*)placeholder_shared + (((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 63; ++ic_chunk_outer_outer) {
    __syncthreads();
    if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 162) {
      if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 24) {
          ((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 648) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)))))[0] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? (int)((int*)((signed char*)placeholder + ((((((ic_chunk_outer_outer * 392) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 196)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 28)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) * 4)) + 360))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
      if (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 288) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((int)threadIdx.z) * 7)) + ((int)threadIdx.y)) < 42) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 4) + ((int)threadIdx.z)) < 6) {
              ((int*)((signed char*)placeholder_shared + (((((((ic_chunk_outer_outer + 1) & 1) * 1152) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + ((((((((((int)blockIdx.y) * 73728) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 72) * 18432)) + (ic_chunk_outer_outer * 288)) + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 72) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 49) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 3) * 4)) + 288))))[0];
          }
        }
      }
    }
    #pragma unroll
    for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
      #pragma unroll
      for (int ic_chunk_inner = 0; ic_chunk_inner < 2; ++ic_chunk_inner) {
        #pragma unroll
        for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(oc_block)] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((((ic_chunk_outer_outer & 1) * 648) + (ic_chunk_inner * 324)) + (((int)threadIdx.y) * 36)) + (kh_inner * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + ((((((((ic_chunk_outer_outer & 1) * 1152) + (((int)threadIdx.z) * 288)) + (ic_chunk_inner * 144)) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(oc_block)]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
    #pragma unroll
    for (int ic_chunk_inner1 = 0; ic_chunk_inner1 < 2; ++ic_chunk_inner1) {
      #pragma unroll
      for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(oc_block1)] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_inner1 * 324) + (((int)threadIdx.y) * 36)) + (kh_inner1 * 36)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 648))))[0], ((int*)((signed char*)placeholder_shared + (((((((((int)threadIdx.z) * 288) + (ic_chunk_inner1 * 144)) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)) + 1152))))[0], compute[(oc_block1)]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax4 = 0; ax4 < 4; ++ax4) {
    ((float*)T_relu)[((((((((int)blockIdx.y) * 784) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)) + ax4))] = max((float)((((float*)placeholder2)[((((((((int)blockIdx.y) * 784) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)) + ax4))] + ((((float)compute[(ax4)]) * 4.186876e-04f) + ((float*)placeholder3)[((((((int)blockIdx.y) * 16) + (((int)threadIdx.z) * 4)) + ax4))]))), (float)(0.000000e+00f));
  }
}

extern "C" __global__ void fused_nn_conv2d_cast_subtract_cast_multiply_add_6_kernel0(void* __restrict__ placeholder, void* __restrict__ placeholder1, void* __restrict__ T_add, void* __restrict__ placeholder2) {
  int compute[16];
  __shared__ int pad_data_shared[192];
  __shared__ int placeholder_shared[288];
  #pragma unroll
  for (int oh_init = 0; oh_init < 4; ++oh_init) {
    #pragma unroll
    for (int oc_block_init = 0; oc_block_init < 4; ++oc_block_init) {
      compute[(((oh_init * 4) + oc_block_init))] = 0;
    }
  }
  if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
    if (((int)threadIdx.z) < 7) {
        ((int*)((signed char*)pad_data_shared + (((((int)threadIdx.z) * 56) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((((int)blockIdx.x) >> 1) * 448) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) - 116))))[0] : (int)(int)0);
    }
  }
  for (int ic_chunk_outer_outer = 0; ic_chunk_outer_outer < 31; ++ic_chunk_outer_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) < 96) {
      if (((int)threadIdx.z) < 7) {
          ((int*)((signed char*)pad_data_shared + ((((((ic_chunk_outer_outer + 1) & 1) * 384) + (((int)threadIdx.z) * 56)) + (((int)threadIdx.x) * 4)))))[0] = (((((1 <= (((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4))) && ((((((int)blockIdx.x) >> 1) * 4) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4)) < 29)) && (1 <= (((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)))) && ((((((int)blockIdx.x) & 1) * 14) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15)) < 29)) ? (int)((int*)((signed char*)placeholder + (((((((ic_chunk_outer_outer * 3136) + ((((int)blockIdx.x) >> 1) * 448)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 4) * 112)) + ((((int)blockIdx.x) & 1) * 56)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 15) * 4)) + 3020))))[0] : (int)(int)0);
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 8) + ((int)threadIdx.z)) < 21) {
            ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + (ic_chunk_outer_outer * 144)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0];
        }
      }
    }
    __syncthreads();
    #pragma unroll
    for (int kw_inner = 0; kw_inner < 3; ++kw_inner) {
      #pragma unroll
      for (int kh_inner = 0; kh_inner < 3; ++kh_inner) {
        #pragma unroll
        for (int oh = 0; oh < 4; ++oh) {
          #pragma unroll
          for (int oc_block = 0; oc_block < 4; ++oc_block) {
            compute[(((oh * 4) + oc_block))] = __dp4a(((int*)((signed char*)pad_data_shared + (((((((ic_chunk_outer_outer & 1) * 384) + (oh * 64)) + (kh_inner * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner * 4)))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner * 48)) + (kw_inner * 16)) + (oc_block * 4)))))[0], compute[(((oh * 4) + oc_block))]);
          }
        }
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) < 288) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 8) + ((int)threadIdx.z)) < 21) {
          ((int*)((signed char*)placeholder_shared + ((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 12) * 48) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)))))[0] = ((int*)((signed char*)placeholder1 + (((((((((int)blockIdx.y) * 36864) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) / 36) * 4608)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 112) + (((int)threadIdx.z) * 14)) + ((int)threadIdx.x)) % 36) / 12) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_ax5_outer_fused_outer_outer_outer1 * 28) + (((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) >> 2)) % 3) * 16)) + ((((((int)threadIdx.z) * 14) + ((int)threadIdx.x)) & 3) * 4)) + 4464))))[0];
      }
    }
  }
  __syncthreads();
  #pragma unroll
  for (int kw_inner1 = 0; kw_inner1 < 3; ++kw_inner1) {
    #pragma unroll
    for (int kh_inner1 = 0; kh_inner1 < 3; ++kh_inner1) {
      #pragma unroll
      for (int oh1 = 0; oh1 < 4; ++oh1) {
        #pragma unroll
        for (int oc_block1 = 0; oc_block1 < 4; ++oc_block1) {
          compute[(((oh1 * 4) + oc_block1))] = __dp4a(((int*)((signed char*)pad_data_shared + ((((((oh1 * 64) + (kh_inner1 * 64)) + (((int)threadIdx.x) * 4)) + (kw_inner1 * 4)) + 384))))[0], ((int*)((signed char*)placeholder_shared + (((((((int)threadIdx.z) * 144) + (kh_inner1 * 48)) + (kw_inner1 * 16)) + (oc_block1 * 4)))))[0], compute[(((oh1 * 4) + oc_block1))]);
        }
      }
    }
  }
  #pragma unroll
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 4; ++ax2_inner_inner_inner) {
    #pragma unroll
    for (int ax4 = 0; ax4 < 4; ++ax4) {
      ((float*)T_add)[((((((((((int)blockIdx.y) * 25088) + (((int)threadIdx.z) * 3136)) + ((((int)blockIdx.x) >> 1) * 448)) + (ax2_inner_inner_inner * 112)) + ((((int)blockIdx.x) & 1) * 56)) + (((int)threadIdx.x) * 4)) + ax4))] = ((((float)compute[(((ax2_inner_inner_inner * 4) + ax4))]) * 9.612082e-05f) + ((float*)placeholder2)[((((((int)blockIdx.y) * 32) + (((int)threadIdx.z) * 4)) + ax4))]);
    }
  }
}

extern "C" __global__ void fused_layout_transform_nn_batch_flatten_kernel0(void* __restrict__ tensor, void* __restrict__ placeholder) {
  if (((int)threadIdx.x) < 512) {
    ((float*)tensor)[(((int)threadIdx.x))] = ((float*)placeholder)[(((int)threadIdx.x))];
  }
}

